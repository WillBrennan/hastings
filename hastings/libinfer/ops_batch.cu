#include "hip/hip_runtime.h"
#include <glog/logging.h>

#include "hastings/libinfer/cu_tensor.h"
#include "hastings/libinfer/ops_batch.h"

namespace libinfer {
namespace detail {
__global__ void cropKernel(CudaTensor<const unsigned char, Ordering::NHWC> cu_image_bgr, Crop* cu_crops,
                           const int num_crops, CudaTensor<unsigned char, Ordering::NHWC> cu_image_crops) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int nc = blockIdx.z * blockDim.z + threadIdx.z;
    const int n = nc / cu_image_crops.c;
    const int c = nc % cu_image_crops.c;

    const int crop_w = cu_image_crops.w;
    const int crop_h = cu_image_crops.h;

    if (x >= crop_w || y >= crop_h || c >= cu_image_crops.c || n >= num_crops) {
        return;
    }

    const auto& crop = cu_crops[n];

    const float bgr_x = crop.tl[0] + (x * crop.shape[0]) / crop_w;
    const float bgr_y = crop.tl[1] + (y * crop.shape[1]) / crop_h;

    const bool inImageX = bgr_x >= 0.0 && bgr_x < cu_image_bgr.w;
    const bool inImageY = bgr_y >= 0.0 && bgr_y < cu_image_bgr.h;
    const bool inImage = inImageX && inImageY;

    if (!inImage) {
        cu_image_crops(n, c, y, x) = 127;
        return;
    }

    const int x0 = int(bgr_x);
    const int y0 = int(bgr_y);
    const int x1 = min(x0 + 1, cu_image_bgr.w - 1);
    const int y1 = min(y0 + 1, cu_image_bgr.h - 1);

    const float v00 = cu_image_bgr(0, c, y0, x0);
    const float v01 = cu_image_bgr(0, c, y0, x1);
    const float v10 = cu_image_bgr(0, c, y1, x0);
    const float v11 = cu_image_bgr(0, c, y1, x1);

    const float interp0 = v00 + (v01 - v00) * (bgr_x - x0);
    const float interp1 = v10 + (v11 - v10) * (bgr_x - x0);
    cu_image_crops(n, c, y, x) = interp0 + (interp1 - interp0) * (bgr_y - y0);
}
}  // namespace detail

void cropFromImage(const Tensor& image_bgr, const std::vector<Crop>& crops, const int maxCrops, const int cropWidth,
                   Tensor& crops_image) {
    CHECK(image_bgr.type() == Type::UINT8) << "images must be uint8";
    CHECK(image_bgr.device() == Device::CUDA) << "images must be on CUDA";
    CHECK(image_bgr.shape().order == Ordering::NHWC) << "images must be NHWC";

    const auto crops_shape = Shape{Ordering::NHWC, maxCrops, 3, cropWidth, cropWidth};
    if (crops_image.shape() != crops_shape) {
        crops_image = Tensor(crops_shape, Device::CUDA, Type::UINT8);
    }

    const auto cu_image_bgr = detail::CudaTensor<const unsigned char, Ordering::NHWC>(image_bgr);
    auto cu_image_crops = detail::CudaTensor<unsigned char, Ordering::NHWC>(crops_image);

    Crop* cu_crops = nullptr;
    const auto numCrops = std::min(maxCrops, int(crops.size()));
    const auto numBytes = sizeof(Crop) * numCrops;

    hipMalloc(&cu_crops, numBytes);
    hipMemcpy(cu_crops, crops.data(), numBytes, hipMemcpyDefault);

    hipDeviceSynchronize();

    dim3 threadsPerBlock(32, 32, 1);
    dim3 numBlocks((cu_image_crops.w + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (cu_image_crops.h + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (cu_image_crops.c * cu_image_crops.n + threadsPerBlock.z - 1) / threadsPerBlock.z);

    detail::cropKernel<<<numBlocks, threadsPerBlock>>>(cu_image_bgr, cu_crops, numCrops, cu_image_crops);
    hipDeviceSynchronize();  // Synchronize to catch errors
}
}  // namespace libinfer