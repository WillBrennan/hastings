#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <glog/logging.h>

#include "hastings/libinfer/cu_tensor.h"
#include "hastings/libinfer/ops.h"

namespace libinfer {
namespace detail {
__device__ float sigmoid(const float x) { return 1.0f / (1.0f + __expf(-x)); }

__device__ void subPixelPeak(const CudaTensor<const float, Ordering::NCHW> logits, const int y, const int x, Peak& peak) {
    const auto x1 = logits(peak.n, peak.c, y, x);
    const auto y1 = x1;

    const auto x0 = logits(peak.n, peak.c, y, x - 1);
    const auto x2 = logits(peak.n, peak.c, y, x + 1);

    const auto y0 = logits(peak.n, peak.c, y - 1, x);
    const auto y2 = logits(peak.n, peak.c, y + 1, x);

    const auto epsilon = 1e-6f;
    auto dx = 0.5 * (x0 - x2) / (x0 + x2 - 2.0f * x1 + epsilon);
    auto dy = 0.5 * (y0 - y2) / (y0 + y2 - 2.0f * y1 + epsilon);

    dx = fmaxf(-0.5f, fminf(0.5, dx));
    dy = fmaxf(-0.5f, fminf(0.5, dy));
}

template <int num_c>
__global__ void preprocessKernel(const CudaTensor<const unsigned char, Ordering::NHWC> image, const Normalize<num_c> norm,
                                 CudaTensor<float, Ordering::NCHW> output) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int n = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= image.w || y >= image.h || n >= image.n) {
        return;
    }

    for (int c = 0; c < num_c; ++c) {
        const auto value = float(image(n, c, y, x)) / 255.0f;
        output(n, c, y, x) = (value - norm.mean[c]) / norm.std[c];
    }
}

__global__ void sigmoidKernel(detail::CudaTensor<float, Ordering::NCHW> logits) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= logits.w || y >= logits.h) {
        return;
    }

    for (int i = 0; i < logits.n; ++i) {
        for (int j = 0; j < logits.c; ++j) {
            logits(i, j, y, x) = sigmoid(logits(i, j, y, x));
        }
    }
}

__global__ void maskFromProbsKernel(const CudaTensor<const float, Ordering::NCHW> probs, CudaTensor<unsigned char, Ordering::NHWC> image) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int n = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= image.w || y >= image.h || n >= image.n) {
        return;
    }

    for (int c = 0; c < probs.c; ++c) {
        image(n, c, y, x) = std::uint8_t(255.0 * probs(n, c, y, x));
    }
}

__global__ void colorByProbsKernel(const CudaTensor<const float, Ordering::NCHW> probs, CudaTensor<unsigned char, Ordering::NHWC> image) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int n = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= image.w || y >= image.h || n >= image.n) {
        return;
    }

    float color[3] = {0, 0, 0};
    const unsigned char colors[6][3] = {{255, 0, 0}, {0, 255, 0}, {0, 0, 255}, {255, 255, 0}, {255, 0, 255}, {0, 255, 255}};

    for (int c = 0; c < probs.c; ++c) {
        const auto prob = probs(n, c, y, x);
        const auto& selected_color = colors[c % 6];
        color[0] += prob * selected_color[0];
        color[1] += prob * selected_color[1];
        color[2] += prob * selected_color[2];
    }

    for (int c = 0; c < 3; ++c) {
        image(n, c, y, x) = 0.5f * image(n, c, y, x) + 0.5f * color[c];
    }
}

template <class Fn>
__global__ void findPeaksKernel(const CudaTensor<const float, Ordering::NCHW> logits, const float threshold, int* num_peaks,
                                const int max_num_peaks, Fn fn) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x + 1;
    const int y = blockIdx.y * blockDim.y + threadIdx.y + 1;
    const int nc = blockIdx.z * blockDim.z + threadIdx.z;

    const int n = nc / logits.c;
    const int c = nc % logits.c;

    const int max_w = logits.w - 1;
    const int max_h = logits.h - 1;

    if (x >= max_w || y >= max_h) {
        return;
    }

    const auto value = logits(n, c, y, x);

    bool is_peak = value >= threshold;
    is_peak &= logits(n, c, y - 1, x - 1) < value;
    is_peak &= logits(n, c, y - 1, x) < value;
    is_peak &= logits(n, c, y - 1, x + 1) < value;

    is_peak &= logits(n, c, y, x - 1) < value;
    is_peak &= logits(n, c, y, x + 1) < value;

    is_peak &= logits(n, c, y + 1, x - 1) < value;
    is_peak &= logits(n, c, y + 1, x) < value;
    is_peak &= logits(n, c, y + 1, x + 1) < value;

    if (is_peak) {
        const int peak_idx = atomicAdd(num_peaks, 1);

        if (peak_idx < max_num_peaks) {
            fn(peak_idx, n, c, y, x, value);
        }
    }
}

struct PeakHelper {
    detail::CudaTensor<const float, Ordering::NCHW> cu_logits;
    Peak* cu_peaks = nullptr;

    __device__ void operator()(const int peak_idx, const int n, const int c, const int y, const int x,
                               const float value) __restrict__ const {
        Peak peak = {n, c, float(x), float(y), value};
        detail::subPixelPeak(cu_logits, y, x, peak);
        cu_peaks[peak_idx] = peak;
    }
};

struct OAPHelper {
    detail::CudaTensor<const float, Ordering::NCHW> cu_logits;
    detail::CudaTensor<const float, Ordering::NCHW> cu_shapes;
    Detection* cu_detections = nullptr;

    __device__ void operator()(const int peak_idx, const int n, const int c, const int y, const int x,
                               const float value) __restrict__ const {
        const auto w = cu_shapes(n, 1, y, x);
        const auto h = cu_shapes(n, 0, y, x);

        Detection peak = {n, c, float(x), float(y), value, w, h};
        detail::subPixelPeak(cu_logits, y, x, peak);

        peak.x *= 8;
        peak.y *= 8;

        cu_detections[peak_idx] = peak;
    }
};

template <class Fn>
int findPeaks(const Tensor& logits, const float min_confidence, const std::size_t max_peaks, const Fn helper) {
    CHECK(logits.type() == Type::FLOAT32) << "logits must be float32";
    CHECK(logits.device() == Device::CUDA) << "logits must be on CUDA";
    CHECK(logits.shape().order == Ordering::NCHW) << "logits must be NCHW";

    int* num_peaks_d = nullptr;
    hipMalloc(&num_peaks_d, sizeof(int));
    hipMemset(num_peaks_d, 0, sizeof(int));

    auto cu_logits = detail::CudaTensor<const float, Ordering::NCHW>(logits);

    dim3 threadsPerBlock(32, 32, 1);
    dim3 numBlocks((cu_logits.w + threadsPerBlock.x - 1) / threadsPerBlock.x, (cu_logits.h + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (cu_logits.n * cu_logits.c + threadsPerBlock.z - 1) / threadsPerBlock.z);

    detail::findPeaksKernel<Fn><<<numBlocks, threadsPerBlock>>>(cu_logits, min_confidence, num_peaks_d, max_peaks, helper);
    hipDeviceSynchronize();

    int num_peaks_h = 0;
    hipMemcpy(&num_peaks_h, num_peaks_d, sizeof(int), hipMemcpyKind::hipMemcpyDefault);
    hipDeviceSynchronize();

    return num_peaks_h;
}

}  // namespace detail

std::ostream& operator<<(std::ostream& stream, const Peak& peak) {
    stream << "{ n: " << peak.n << " c: " << peak.c << " y: " << peak.y << " x: " << peak.x << " }";
    return stream;
}

std::ostream& operator<<(std::ostream& stream, const Detection& det) {
    stream << "{ peak: " << Peak(det) << " w: " << det.w << " h: " << det.h << " }";
    return stream;
}

template <std::size_t c>
void preprocessImpl(const Tensor& images, const Normalize<c> norm, Tensor& output) {
    CHECK(images.type() == Type::UINT8) << "images must be uint8";
    CHECK(images.device() == Device::CUDA) << "images must be on CUDA";
    CHECK(images.shape().order == Ordering::NHWC) << "images must be NHWC";
    CHECK_EQ(images.shape().c, c) << "image has the wrong number of channels";

    if (output.device() == Device::Empty) {
        auto output_shape = images.shape();
        output_shape.order = Ordering::NCHW;
        output = Tensor(output_shape, Device::CUDA, Type::FLOAT32);
    }

    const auto cu_image = detail::CudaTensor<const unsigned char, Ordering::NHWC>(images);
    auto cu_output = detail::CudaTensor<float, Ordering::NCHW>(output);

    dim3 threadsPerBlock(32, 32, 1);
    dim3 numBlocks((cu_image.w + threadsPerBlock.x - 1) / threadsPerBlock.x, (cu_image.h + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (cu_image.n + threadsPerBlock.z - 1) / threadsPerBlock.z);

    detail::preprocessKernel<c><<<numBlocks, threadsPerBlock>>>(cu_image, norm, cu_output);
}

void preprocess(const Tensor& images, const Normalize<1> norm, Tensor& output) { preprocessImpl<1>(images, norm, output); }

void preprocess(const Tensor& images, const Normalize<3> norm, Tensor& output) { preprocessImpl<3>(images, norm, output); }

void sigmoid(Tensor& logits) {
    CHECK(logits.type() == Type::FLOAT32) << "logits must be float32";
    CHECK(logits.device() == Device::CUDA) << "logits must be on CUDA";
    CHECK(logits.shape().order == Ordering::NCHW) << "logits must be NCHW";

    const auto& shape = logits.shape();
    auto cu_logits = detail::CudaTensor<float, Ordering::NCHW>(logits);

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((shape.w + threadsPerBlock.x - 1) / threadsPerBlock.x, (shape.h + threadsPerBlock.y - 1) / threadsPerBlock.y);

    detail::sigmoidKernel<<<numBlocks, threadsPerBlock>>>(cu_logits);

    hipDeviceSynchronize();
}

void maskFromProbs(const Tensor& probs, Tensor& image) {
    CHECK(image.type() == Type::UINT8) << "image must be uint8";
    CHECK(image.device() == Device::CUDA) << "image must be on CUDA";
    CHECK(image.shape().order == Ordering::NHWC) << "image must be NHWC";
    CHECK(probs.type() == Type::FLOAT32) << "images must be uint8";
    CHECK(probs.device() == Device::CUDA) << "images must be on CUDA";
    CHECK(probs.shape().order == Ordering::NCHW) << "images must be NHWC";

    auto cu_image = detail::CudaTensor<unsigned char, Ordering::NHWC>(image);
    const auto cu_probs = detail::CudaTensor<const float, Ordering::NCHW>(probs);

    CHECK_EQ(cu_image.c, cu_probs.c);

    dim3 threadsPerBlock(32, 32, 1);
    dim3 numBlocks((cu_probs.w + threadsPerBlock.x - 1) / threadsPerBlock.x, (cu_probs.h + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (cu_probs.n + threadsPerBlock.z - 1) / threadsPerBlock.z);

    detail::maskFromProbsKernel<<<numBlocks, threadsPerBlock>>>(cu_probs, cu_image);
}

void colorByProbs(const Tensor& probs, Tensor& image) {
    CHECK(image.type() == Type::UINT8) << "image must be uint8";
    CHECK(image.device() == Device::CUDA) << "image must be on CUDA";
    CHECK(image.shape().order == Ordering::NHWC) << "image must be NHWC";
    CHECK(probs.type() == Type::FLOAT32) << "images must be uint8";
    CHECK(probs.device() == Device::CUDA) << "images must be on CUDA";
    CHECK(probs.shape().order == Ordering::NCHW) << "images must be NHWC";

    auto cu_image = detail::CudaTensor<unsigned char, Ordering::NHWC>(image);
    const auto cu_probs = detail::CudaTensor<const float, Ordering::NCHW>(probs);

    dim3 threadsPerBlock(32, 32, 1);
    dim3 numBlocks((cu_probs.w + threadsPerBlock.x - 1) / threadsPerBlock.x, (cu_probs.h + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (cu_probs.n + threadsPerBlock.z - 1) / threadsPerBlock.z);

    detail::colorByProbsKernel<<<numBlocks, threadsPerBlock>>>(cu_probs, cu_image);
}

void peakFinding(const Tensor& logits, const float min_confidence, const std::size_t max_peaks, std::vector<Peak>& peaks) {
    Peak* cu_peaks = nullptr;
    hipMalloc(&cu_peaks, sizeof(Peak) * max_peaks);

    auto cu_logits = detail::CudaTensor<const float, Ordering::NCHW>(logits);

    const auto helper = detail::PeakHelper{cu_logits, cu_peaks};
    const int num_peaks = detail::findPeaks(logits, min_confidence, max_peaks, helper);

    peaks.resize(num_peaks);
    hipMemcpy(peaks.data(), cu_peaks, sizeof(Peak) * num_peaks, hipMemcpyKind::hipMemcpyDefault);

    hipDeviceSynchronize();
}

void objectsAsPoints(const Tensor& logits, const Tensor& shapes, const float min_confidence, const std::size_t max_detections,
                     std::vector<Detection>& detections) {
    CHECK(shapes.type() == Type::FLOAT32) << "shapes must be float32";
    CHECK(shapes.device() == Device::CUDA) << "shapes must be on CUDA";
    CHECK(shapes.shape().order == Ordering::NCHW) << "shapes must be NCHW";

    Detection* cu_detections = nullptr;
    hipMalloc(&cu_detections, sizeof(Detection) * max_detections);

    auto cu_logits = detail::CudaTensor<const float, Ordering::NCHW>(logits);
    auto cu_shapes = detail::CudaTensor<const float, Ordering::NCHW>(shapes);
    const auto helper = detail::OAPHelper{cu_logits, cu_shapes, cu_detections};

    const int num_peaks = detail::findPeaks(logits, min_confidence, max_detections, helper);

    detections.resize(num_peaks);
    hipMemcpy(detections.data(), cu_detections, sizeof(Detection) * num_peaks, hipMemcpyKind::hipMemcpyDefault);

    hipDeviceSynchronize();
}
}  // namespace libinfer